#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#include <thrust/device_vector.h>
#include <thrust/inner_product.h>

#include "mpi.h"

#define N_ROUND 100

static int mpi_size = 0, mpi_rank = 0;

#define CHECK(cond, errmsg) \
  if (!(cond)) { \
    fprintf(stderr, "[rank%d](%s:%d) %s\n", mpi_rank, __FILE__, __LINE__, errmsg); \
    exit(1); \
  }

double dot_product(long n, double *x, double *y) {
  thrust::device_vector<double> d_x(x, x + n);
  thrust::device_vector<double> d_y(y, y + n);
  return thrust::inner_product(d_x.begin(), d_x.end(), d_y.begin(), 0.0);
}

int main() {
  MPI_Init(NULL, NULL);
  MPI_Comm_size(MPI_COMM_WORLD, &mpi_size);
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);

  int cuda_num_dev;
  CHECK(hipGetDeviceCount(&cuda_num_dev) == hipSuccess, "Failed to invoke hipGetDeviceCount.");
  CHECK(hipSetDevice(mpi_rank % cuda_num_dev) == hipSuccess, "Failed to invoke hipSetDevice.");

  long n_elems, n_ranks;
  double *arr_x, *arr_y;

  // Read data from file
  char filename[1024];
  snprintf(filename, 1024, "input.%d.bin", mpi_rank);
  FILE *f = fopen(filename, "r");
  CHECK(f != NULL, "Failed to open file.");

  CHECK(fread(&n_elems, sizeof(n_elems), 1, f) != sizeof(n_elems), "Failed to read n_elems.");
  CHECK(fread(&n_ranks, sizeof(n_ranks), 1, f) != sizeof(n_ranks), "Failed to read n_ranks.");
  CHECK(n_ranks == mpi_size, "Number of partitions (input file) mismatches number of mpi ranks.");

  size_t arr_size = sizeof(double) * n_elems;
  arr_x = (double*)malloc(arr_size); // Dynamically allocate memory
  arr_y = (double*)malloc(arr_size);
  CHECK(fread(arr_x, arr_size, 1, f) != arr_size, "Failed to read array X.");
  CHECK(fread(arr_y, arr_size, 1, f) != arr_size, "Failed to read array Y.");

  // Compute
  double elapsed_time = 0;
  double partial_res, res;
  for (int i = 1; i <= N_ROUND; i++) {
    double start, stop;
    MPI_Barrier(MPI_COMM_WORLD); // Synchronize all mpi processes
    start = MPI_Wtime() * 1e6; // Unit: sec -> micro sec

    partial_res = dot_product(n_elems, arr_x, arr_y); // Compute Kernel
    MPI_Allreduce(&partial_res, &res, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD); // Aggregate result (with implicit synchronization)

    stop = MPI_Wtime() * 1e6;
    elapsed_time += stop - start;
    printf("[rank%d] Round: %d, Partial Result: %lf, Result: %lf, Average Elapsed Time: %.0lfus\n",
      mpi_rank, i, partial_res, res, elapsed_time / i);
  }
  
  printf("\n\n[rank%d] Result: %lf, Average Elapsed Time: %.0lfus\n",
    mpi_rank, res, elapsed_time / N_ROUND);

  // Finalize & Clean up
  fclose(f);
  free(arr_x);
  free(arr_y);

  MPI_Finalize();
  return 0;
}